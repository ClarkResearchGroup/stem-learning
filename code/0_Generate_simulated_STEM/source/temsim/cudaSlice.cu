#include "hip/hip_runtime.h"
/*      
    *** cudaSlice.cu ***   

------------------------------------------------------------------------
Copyright 2018-2019 Earl J. Kirkland


This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.

---------------------- NO WARRANTY ------------------
THIS PROGRAM IS PROVIDED AS-IS WITH ABSOLUTELY NO WARRANTY
OR GUARANTEE OF ANY KIND, EITHER EXPRESSED OR IMPLIED,
INCLUDING BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF
MERCHANABILITY AND FITNESS FOR A PARTICULAR PURPOSE.
IN NO EVENT SHALL THE AUTHOR BE LIABLE
FOR DAMAGES RESULTING FROM THE USE OR INABILITY TO USE THIS
PROGRAM (INCLUDING BUT NOT LIMITED TO LOSS OF DATA OR DATA
BEING RENDERED INACCURATE OR LOSSES SUSTAINED BY YOU OR
THIRD PARTIES OR A FAILURE OF THE PROGRAM TO OPERATE WITH
ANY OTHER PROGRAM). 

-----------------------------------------------------------------------------

  cuda subroutines:

	cmplPixMul()     : complex pix mul with shift
	cmplVecMul()     : complex vector mul
	cuAtompot()      : calculate atomic potential of one slice
	cuBWlimit()      : bandwidth limit
	cuFreq()         : calculate FFT frequencies
	cuPhasegrating() : phase grating
	integCBED()      : integrate ADF detector
	magSqPix()       : form sq. magnitude of pix
	probeShift()     : shift probe in FFT space
	zeroDbleArray()  : set double array to zero

  started from autostem_cuda.cu 18-aug-2018 ejk
  add probeShift(), zeroDbleArray(), integCBED() 18-aug-2018 ejk
  add cuAtompot(), cuBWlimit(), cuPhasegrating()  27-sep-2018 ejk 
  add full cache (for all NZMAX) to cuAtompot() 15-sep-2019 ejk

    this file is formatted for a TAB size of 4 characters 
*/


#ifndef CUDASUBS_INCLUDED
#define CUDASUBS_INCLUDED

//==================  extra CUDA stuff ===================


//---------------  cmplPixMul() --------------
//
// CUDA kernel definition for 2D pix mul with trans larger than probe
//     probe is mul. by a subset of trans
//  perform operation probe = probe * trans with offset
//  probe = nxprobe x nyprobe (no bigger than trans)
//  trans = nx x ny  (may be bigger than probe)
//  ixoff, iyoff = offset of probe inside trans ; edges will wrap around
//
__global__ void cmplPixMul( hipfftComplex *trans, hipfftComplex *probe, int nx, int ny,
     int nxprobe, int nyprobe, int ixoff, int iyoff) 
{
    // 2D index into probe array
    int ix = blockDim.x*blockIdx.x + threadIdx.x;  //  range 0 to (nxprobe-1)
    int iy = blockDim.y*blockIdx.y + threadIdx.y;  //  range 0 to (nyprobe-1)

    //  use scratch var so probe[] can be overwriten
    if( (ix < nxprobe) && (iy < nyprobe) ) {

        //  calculate 2D index into trans array
        int ixt = ix + ixoff;
        if( ixt >= nx ) ixt = ixt - nx;
        else if( ixt < 0 ) ixt = ixt + nx;

        int iyt = iy + iyoff;
        if( iyt >= ny ) iyt = iyt - ny;
        else if( iyt < 0 ) iyt = iyt + ny;

        int it = iyt + ixt*ny;
        int ip = iy + ix*nyprobe;

        float ar, ai, br, bi;
 
        ar = trans[it].x;  // real
        ai = trans[it].y;  // imag
        br = probe[ip].x;
        bi = probe[ip].y;
        probe[ip].x = ar*br - ai*bi;  // real
        probe[ip].y = ar*bi + ai*br;  // imag
    }
}   //  end complPixMul()

//---------------  cmplVecMul() --------------
//
// cuda kernel definition for complex vector mul
//   c = a * b (element by element)
//
__global__ void cmplVecMul( hipfftComplex *a, hipfftComplex *b, hipfftComplex *c, int nmax) 
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    //  use scratch var so c[] can overwrite one of inputs
    if( i < nmax ) {
        float ar, ai, br, bi;
        ar = a[i].x;  // real
        ai = a[i].y;  // imag
        br = b[i].x;
        bi = b[i].y;
        c[i].x = ar*br - ai*bi;
        c[i].y = ar*bi + ai*br;
    }
}  //  end cmplPixMul()

/*---------------  cuAtompot() --------------

  CUDA kernel definition to calculate single layer projected atomic potential

  this is actually no faster than doing the potential in real space on the host
	but save anyway just in case!

  calculate the summation over atoms at one point (kx,ky) in reciprocal space
  
  It is better to sum in reciprocal space to use fine-grain parallelism
  as on a GPU. Every point can then run in parallel without trying to
  access the same point. This is very different than the openMP version
  of trlayer() in autostem.cpp which summs the atomic potential in real space.

  potn[] = nx x ny  output array = half of complex plane for C2R FFT

  x[],y[] = real array of atomic coordinates
  occ[]   = real array of occupancies
  Znum[]  = array of atomic numbers

  spec[ k + 4*iatom] = packed array of x,y,occ,Znum (min. GPU transfers)
						(k=0,1,2,3 for x,y,occ,Znum)
  istart  = starting index of atom coord.
  natom   = number of atoms
  ax, by  = size of transmission function in Angstroms
  kev     = beam energy in keV
  trans   = 2D array to get complex specimen
        transmission function
  nx, ny  = dimensions of transmission functions
  *phirms = average phase shift of projected atomic potential
  *nbeams = will get number of Fourier coefficients
  k2max   = square of max k = bandwidth limit
  fparams[] = scattering factor parameters

  scale = mm0 * wavelength (put here for comparison to original trlayer()

	repeat scaling from mulslice.cpp
		mm0 = 1.0F + v0/511.0F;
		wavlen = (float) wavelength( v0 );
		scale = wavlen * mm0;

*/

__global__ void cuAtompot( hipfftComplex *potn, 
	 float spec[], int natom,  int istart,
    const float ax, const float by, const float kev,
    const int nx, const int ny,
	float kx[], float ky[], float kx2[], float ky2[],
    const float k2max, double fparams[], const float scale ) 
{

	// for the atomic scattering factor tables
	const int NPMAX=   12;  // number of parameters for each Z
	const int NZMIN=   1;   // min Z 
	const int NZMAX=   103; // max Z 
	const int nl=3, ng=3;   // number of Lorenzians and Gaussians

	int i, j, iatom, Z, nymid;
	float k2, x, y, occ;
	double sumf, sumr, sumi, w;
	double fe[NZMAX+1];  // scatt. factor
	const double twopi = 6.283185307;

    // 2D index into trans array
    int ix = blockDim.x*blockIdx.x + threadIdx.x;  //  range 0 to (nxprobe-1)
    int iy = blockDim.y*blockIdx.y + threadIdx.y;  //  range 0 to (nyprobe-1)

	nymid = 1 + ny/2;

    //  valid trans[] index
    if( (ix < nx) && (iy < nymid) ) {

		// init to something in case it fails later
		potn[ iy + ix*nymid].x = 0.0F;  // real part
		potn[ iy + ix*nymid].y = 0.0F;  // imag part

		k2 = kx2[ix]+ky2[iy];

		if( k2 < k2max ) {
            
            // init scatt. factor to <0 to indicate not yet valid
            //   - should always be > 0.0 for real values
            for( j=0; j<(NZMAX+1); j++) fe[j] = -100.0;

			sumr = sumi = 0.0;
			for( iatom=istart; iatom<(istart+natom); iatom++) {

				Z = (int) ( spec[3 + 4*iatom] + 0.2); // round up to avoid truncation
				if( (Z<NZMIN) || (Z>NZMAX) ) return;

				// save old values in a look up table for repeated Z 
				//  - don't repeat sum - speeds thing up a lot  
				if( fe[Z] <  0.0 ) {
					sumf = 0.0;

					// Lorenzians - from slicelib.cpp  
					for( i=0; i<2*nl; i+=2 )
						sumf += fparams[i + Z*NPMAX]/( k2 + fparams[i+1 + Z*NPMAX] );
							 // mimic fparams[Z][i]/( k2 + fparams[Z][i+1] );

					// Gaussians - from slicelib.cpp 
					for( i=2*nl; i<2*(nl+ng); i+=2 )
						sumf += fparams[i + Z*NPMAX]*exp( - k2 * fparams[i+1 + Z*NPMAX] );
							// mimic fparams[Z][i]*exp( - k2 * fparams[Z][i+1] );

					fe[Z] = sumf;  // save scattering factor for this k
				}
				x = spec[ 0 + 4*iatom];
				y = spec[ 1 + 4*iatom];
				occ = spec[ 2 + 4*iatom];
				w = twopi * ( kx[ix]*x + ky[iy]*y );

				sumr += fe[Z] * cos( -w ) * occ;
				sumi += fe[Z] * sin( -w ) * occ;

			}  //end for( iatom...)

			potn[ iy + ix*nymid].x = scale * sumr;  // real part
			potn[ iy + ix*nymid].y = scale * sumi;  // imag part

		}  // end if( k2 < ....

	}  //  end if((ix<nx)....

	return;

}   //  end cuAtompot()

/*---------------  cuBWlimit() --------------

  bandwidth limit tran[] - assumed to be in reciprocal space
  and add FFT scale
  
  kx2[],ky2[] = spatial freq. sq.
  k2max = max spatial freq.

*/

__global__ void cuBWlimit( hipfftComplex *trans, 
	float *kx2, float *ky2, float k2max, const int nx, const int ny ) 
{

    // 2D index into trans array
    int ix = blockDim.x*blockIdx.x + threadIdx.x;  //  range 0 to (nxprobe-1)
    int iy = blockDim.y*blockIdx.y + threadIdx.y;  //  range 0 to (nyprobe-1)

    if( (ix < nx) && (iy < ny) ) {
		float scale = 1.0F/( (float) (nx*ny) );

        float k2 = kx2[ix] + ky2[iy];

        if( k2 > k2max ) {
        	trans[iy + ix*ny].x = 0.0F;
        	trans[iy + ix*ny].y = 0.0F;
        } else {
        	trans[iy + ix*ny].x *= scale;  //  fix FFT scale
        	trans[iy + ix*ny].y *= scale;
		}
        
    }  // end if( ix< nx...
    
}  //  end cuBWlimit()

/*---------------  cuFreq() --------------
//
// cuda kernel definition to calculate spatial freq.
//   
    ko[n]  = real array to get spatial frequencies
    ko2[n] = real array to get k[i]*k[i] 
    nk     = integer number of pixels
    ak     = real full scale size of image in pixels
*/
__global__ void cuFreq( float ko[], float ko2[], int nk, float ak ) 
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    //  check for valid thread index
    if( i < nk) {

		int imid = (int) ( nk/2.0 + 0.5);   /* when nk may not be 2^m */

        if ( i > imid ) {
            ko[i]  = ((float)(i-nk)) / ((float)ak);
        } else {
            ko[i]  = ((float)i) / ((float)ak);
        }
        ko2[i] = ko[i] * ko[i];
    }
}  //  end cuFreq()

/*---------------  cuPhasegrating() --------------

  Start with the atomic potential from cuAtompot() after inv. FFT
  and convert to the transmission function as in a phase grating calculation
  - assume its scaled to a phase already
*/

__global__ void cuPhasegrating( float * potnR, hipfftComplex *trans, 
    const int nx, const int ny ) 
{

    // 2D index into trans array
    int ix = blockDim.x*blockIdx.x + threadIdx.x;  //  range 0 to (nxprobe-1)
    int iy = blockDim.y*blockIdx.y + threadIdx.y;  //  range 0 to (nyprobe-1)

    if( (ix < nx) && (iy < ny) ) {
         
		//  atomic potential
		double vz= potnR[iy + ix*ny];
		trans[iy + ix*ny].x = (float) cos(vz);  // real
        trans[iy + ix*ny].y = (float) sin(vz);  // imag

    }  // end if( (ix<nx)....
    
}   //  end cuPhasegrating()


//---------------  integCBED() --------------
//
// CUDA kernel definition to integrate STEM detector active regions
//
//  remember:
//      [1] many threads cannot access the same sumation variable at
//           one time so sum along only one direction at a time (into a 1D array)
//				- complete the last sum in 1D on the host
//      [2] many points will not be on the active portion of the detector
//             so there is less competition among threads than it might seem
//
//  cbed = input nx x ny float CBED pix = |cpix|^2
//  sums = oout double[nx]  to get sum |cpix|^2 along iy
//  nx, ny = size of cbed
//  collectorMode = detector type
//  kxp[],kyp[] = spatial freq.
//  kxp2[],kyp2[] = spatial freq. sq.
//  k2min, k2max = detector range in polar direction
//  phimin, phimax = detector range in azimuthal direction
//
__global__ void integCBED( double *sums, float *cbed, int nx, int ny,
    int collectorMode, float *kxp, float *kyp, float *kxp2, float *kyp2,
    float k2min, float k2max, float phiMin, float phiMax ) 
{
    // modes of collector
    enum{ ADF=0, CONFOCAL=1, ADF_SEG=2, TOTAL=3};  //  no confocal here yet
    
    // 2D index into fpix array
    int ix = blockDim.x*blockIdx.x + threadIdx.x;  //  range 0 to (nx-1)
    int iy = blockDim.y*blockIdx.y + threadIdx.y;  //  range 0 to (ny-1)

    if( (ix < nx) && (iy < ny) ) {

        //  calculate 2D index into cbed array
        int it = iy + ix*ny;
        float k2, phi;
        
        k2 = kxp2[ix] + kyp2[iy];

        //  use atomicSum() so only one thread can access sums[] at a time
		//    - need nvcc compiler option "-arch=sm_61" for atomicAdd()
        if( ADF == collectorMode ) {
            if( (k2 >= k2min ) && (k2 < k2max ) ) atomicAdd( &(sums[iy]), (double)cbed[it] );
        } else if( ADF_SEG == collectorMode ) {
            phi = atan2( kyp[iy], kxp[ix] ); 
            if( (k2 >= k2min ) && (k2 < k2max)  
				&& (phi >= phiMin) && (phi< phiMax) )
						atomicAdd( &(sums[iy]), (double)cbed[it] );
        } else if( TOTAL == collectorMode ) {
				atomicAdd( &(sums[iy]), (double)cbed[it] );
        }

    }  // end if( (ix < nx)....
    
}   //  end integCBED()


//---------------  magSqPix() --------------
//
// CUDA kernel definition for 2D pix complex to magnitude
//     take square magnitude on GPU to
// 
//  cpix = nx x ny complex
//  fpix = nx x ny  float = |cpix|^2
//  nx, ny = size of both pix
//
__global__ void magSqPix( float *fpix, hipfftComplex *cpix, int nx, int ny ) 
{
    // 2D index into both arrays
    int ix = blockDim.x*blockIdx.x + threadIdx.x;  //  range 0 to (nx-1)
    int iy = blockDim.y*blockIdx.y + threadIdx.y;  //  range 0 to (ny-1)

    if( (ix < nx) && (iy < ny) ) {

        int it = iy + ix*ny;  //  calculate 2D index into trans array
        float ar, ai;
 
        ar = cpix[it].x;  // real
        ai = cpix[it].y;  // imag
        fpix[it] = ar*ar + ai*ai;
    }
}   //  end magSqPix()

//---------------  probeShift() --------------
//
//  CUDA kernel definition for 2D probe shift in FT space
//  perform operation probe *= exp( 2*pi*i * x * k) with offset
//
//  prb0 = input nx x ny complex
//  prbs = output gets prb0 shifted by (xs,ys)
//  xs,ys = amount to shift
//  nx, ny = size of both pix
//  kx[], ky[] = arrays of spatial frequencies
//
__global__ void probeShift( hipfftComplex *prbs, hipfftComplex *prb0, int nx, int ny,
    float xs, float ys, float *kx, float *ky ) 
{
    // 2D index into both arrays
    int ix = blockDim.x*blockIdx.x + threadIdx.x;  //  range 0 to (nx-1)
    int iy = blockDim.y*blockIdx.y + threadIdx.y;  //  range 0 to (ny-1)

    if( (ix < nx) && (iy < ny) ) {

        int it = iy + ix*ny;  //  calculate 2D index into arrays
        
        double ar, ai, br, bi;
        double w = 6.283185307*(xs*kx[ix] + ys*ky[iy]);
 
        ar = prb0[it].x;  // real
        ai = prb0[it].y;  // imag
        br = cos( w );
        bi = sin( w );
        
        prbs[it].x = (float) (ar*br - ai*bi);  // real
        prbs[it].y = (float) (ar*bi + ai*br);  // imag
    }
}   //  end probeShift()


//---------------  zeroDbleArray() --------------
//
// CUDA kernel definition to zero a double array
// 
//  a[nmax] = double array
//  nmax = size of array
//
__global__ void zeroDbleArray( double *a, int nmax ) 
{
    // index into array
    int i = blockDim.x*blockIdx.x + threadIdx.x;  //  range 0 to (nmax-1)

    if( i < nmax ) a[i] = 0.0;

}   //  end zeroDbleArray()

#endif
